#include "hip/hip_runtime.h"
#include "cudaconfidencemaps_cuda.h"
#include "cudautils.h"

#include <cusp/blas.h>
#include <cusp/dia_matrix.h>
#include <cusp/krylov/cg.h>
#include <cusp/precond/diagonal.h>

typedef cusp::dia_matrix<int, float, cusp::host_memory> CuspHostDiaMatrix;
typedef cusp::dia_matrix<int, float, cusp::device_memory> CuspDeviceDiaMatrix;
typedef cusp::array1d<float, cusp::host_memory> CuspHostVector;
typedef cusp::array1d<float, cusp::device_memory> CuspDeviceVector;
typedef cusp::array1d<unsigned char, cusp::device_memory> CuspDeviceByteVector;

/* The implicit graph created by this class to solve the confidence maps problem is as follows
 *
 *   1   1   1   1
 *   |   |   |   |  << These edges have always value 1
 *   #---#---#---#  \
 *   | X | X | X |  |   The graph here is densly connected (8 neighbourhood). The weight
 *   #---#---#---#  |   of all the edges here are computed based on the intensity differences
 *   | X | X | X |  >   of neighbouring pixels. Each node (#) maps to one pixel in the image.
 *   #---#---#---#  |   Overall, the equation system Lx = b represents a diffusion problem.
 *   | X | X | X |  |   By construction L is SPD.
 *   #---#---#---#  /
 *   |   |   |   |  << These edges have always value 1
 *   0   0   0   0
 */

namespace campvis {
namespace cuda {

    /**
     * This struct is used as a sort of Private Implementation of the \see CudaConfidenceMapsSystemSolver
     * class (PIMPL idiom). This is necessary, since it is not possible to expose CUSP types, such as
     * vectors and matrices to code that is not compiled using the CUDA compilers
     */
    struct CudaConfidenceMapsSystemGPUData {
        // Data on the host (only solution vector x and matrix L)
        CuspHostDiaMatrix L_h;
        CuspHostVector x_h;

        // Data on the device, a.k.a. GPU (Lx = b)
        CuspDeviceDiaMatrix L_d;
        CuspDeviceVector b_d;
        CuspDeviceVector x_d;

        // Additional data needed to perform the alpha-beta filtering
        CuspDeviceVector abFilterX_d; // Solution estimate
        CuspDeviceVector abFilterV_d; // Velocity
        CuspDeviceVector abFilterR_d; // Residual
        float abFilterAlpha;
        float abFilterBeta;

        // An additional image-sized 8-bit buffer allocated on the gpu that can be used as intermediate
        // storage when needed (e.g. when creating the system of equations on the gpu)
        // This avoids allocations on the GPU on every iteration
        CuspDeviceByteVector imageBuffer_d;

        // Information about the system as well as statistics of the solution
        bool isUpsideDown;
        bool useAlphaBetaFiltering;
        bool use8Neighbourhood; ///< If set to true the full 8-neighbourhood is used, otherwise only the 4-neighbourhood
        int imageWidth;
        int imageHeight;
        int iterationCount;
        float solutionResidualNorm;
        float systemCreationTime;
        float systemSolveTime;
    };

    template <typename ValueType>
    class iteration_monitor : public cusp::default_monitor<ValueType>
    {
        typedef typename cusp::norm_type<ValueType>::type Real;
        typedef cusp::default_monitor<ValueType> super;

    public:
        template <typename Vector>
        iteration_monitor(const Vector& b, size_t iteration_limit = 500)
            : super(b, iteration_limit, 0.0f, 0.0f)
        { }

        template <typename Vector>
        bool finished(const Vector& r)
        {
            // Only if the maximum iteration count has been reached, actually go ahead and
            // compute the error
            if (super::iteration_count() >= super::iteration_limit()) {
                super::r_norm = cusp::blas::nrm2(r);
                return true;
            }

            return false;
        }
    };

    CudaConfidenceMapsSystemSolver::CudaConfidenceMapsSystemSolver()
        : _gpuData(new CudaConfidenceMapsSystemGPUData())
    {
        _gpuData->imageWidth = 0;
        _gpuData->imageHeight = 0;
        _gpuData->solutionResidualNorm = 0.0f;
        _gpuData->systemCreationTime = 0.0f;
        _gpuData->systemSolveTime = 0.0f;
        _gpuData->isUpsideDown = true;
        _gpuData->use8Neighbourhood = true;

        _gpuData->useAlphaBetaFiltering = false;
        _gpuData->abFilterAlpha = 0.125;
        _gpuData->abFilterBeta = 0.250;
    }

    CudaConfidenceMapsSystemSolver::~CudaConfidenceMapsSystemSolver()
    {
        // This also frees all the memory reserved on the GPU
        delete _gpuData;
    }

    void CudaConfidenceMapsSystemSolver::uploadImage(const unsigned char* imageData, int imageWidth, int imageHeight,
                                                     float gradientScaling, float alpha, float beta, float gamma,
                                                     bool use8Neighbourhood, bool isUpsideDown) {
        resizeDataStructures(imageWidth, imageHeight, isUpsideDown, use8Neighbourhood);

        // Measure execution time and record it in the _gpuData datastructure
        CUDAClock clock; clock.start();

        createSystemGPU(imageData, imageWidth, imageHeight, gradientScaling, alpha, beta, gamma, isUpsideDown);

        _gpuData->systemCreationTime = clock.getElapsedMilliseconds();
    }

    void CudaConfidenceMapsSystemSolver::resetSolution() {
        int width = _gpuData->imageWidth;
        int height = _gpuData->imageHeight;
        bool isUpsideDown = _gpuData->isUpsideDown;

        // Compute a linear transition image from white to black and set it as current
        // solution vector
        for (int y = 0; y < height; ++y) {
            for (int x = 0; x < width; ++x) {
                float value;
                if (isUpsideDown) {
                    value = y / (height - 1.0f);
                }
                else {
                    value = 1.0f - y / (height - 1.0f);
                }

                _gpuData->x_h[y * width + x] = value;
            }
        }

        // Uplaod solution vector on the GPU
        _gpuData->x_d = _gpuData->x_h;

        // Prepare data structures for alpha-beta filtering
        _gpuData->abFilterX_d = _gpuData->x_d;
        cusp::blas::fill(_gpuData->abFilterV_d, 0.0f);
    }

    bool CudaConfidenceMapsSystemSolver::alphaBetaFilterEnabled() const {
        return _gpuData->useAlphaBetaFiltering;
    }

    void CudaConfidenceMapsSystemSolver::enableAlphaBetaFilter(bool enabled) {
        _gpuData->useAlphaBetaFiltering = enabled;
    }

    void CudaConfidenceMapsSystemSolver::setAlphaBetaFilterParameters(float alpha, float beta) {
        _gpuData->abFilterAlpha = alpha;
        _gpuData->abFilterBeta = beta;
    }

    // FIXME: Remove errorTolerance parameter
    void CudaConfidenceMapsSystemSolver::solve(int maximumIterations, float errorTolerance) {
        // Measure execution time and record it in the _gpuData datastructure
        CUDAClock clock; clock.start();

        // The solution is computed using Conjugate Gradient with a Diagonal (Jacobi) preconditioner
        iteration_monitor<float> monitor(_gpuData->b_d, maximumIterations);
        cusp::precond::diagonal<float, cusp::device_memory> M(_gpuData->L_d);
        cusp::krylov::cg(_gpuData->L_d, _gpuData->x_d, _gpuData->b_d, monitor, M);
        _gpuData->solutionResidualNorm = monitor.residual_norm();
        _gpuData->iterationCount = static_cast<int>(monitor.iteration_count());

        if (alphaBetaFilterEnabled()) {
            // X' = X' + V'
            // R' = X - X'
            cusp::blas::axpy(_gpuData->abFilterV_d, _gpuData->abFilterX_d, 1.0f);
            cusp::blas::axpby(_gpuData->x_d, _gpuData->abFilterX_d, _gpuData->abFilterR_d, 1.0f, -1.0f);

            // X' = X' + alpha * R'
            // V' = V' + beta * R'
            cusp::blas::axpy(_gpuData->abFilterR_d, _gpuData->abFilterX_d, _gpuData->abFilterAlpha);
            cusp::blas::axpy(_gpuData->abFilterR_d, _gpuData->abFilterV_d, _gpuData->abFilterBeta);

            // Download the smoothed solution to the host
            _gpuData->x_h = _gpuData->abFilterX_d;
        }
        else {
            // Downlaod the actual solution, which has been computed to the host
            _gpuData->x_h = _gpuData->x_d;
        }

        _gpuData->systemSolveTime = clock.getElapsedMilliseconds();
    }

    const float* CudaConfidenceMapsSystemSolver::getSolution(int& width, int& height) {
        width = _gpuData->imageWidth;
        height = _gpuData->imageHeight;
        return thrust::raw_pointer_cast(&_gpuData->x_h[0]);
    }

    int CudaConfidenceMapsSystemSolver::getSolutionIterationCount() const {
        return _gpuData->iterationCount;
    }


    float CudaConfidenceMapsSystemSolver::getSolutionResidualNorm() const {
        return _gpuData->solutionResidualNorm;
    }

    float CudaConfidenceMapsSystemSolver::getSystemCreationTime() const {
        return _gpuData->systemCreationTime;
    }
    
    float CudaConfidenceMapsSystemSolver::getSystemSolveTime() const {
        return _gpuData->systemSolveTime;
    }


    void CudaConfidenceMapsSystemSolver::resizeDataStructures(int imageWidth, int imageHeight, bool isUpsideDown, bool use8Neighbourhood) {
        // If the problem size changed, reset the solution vector, as well as all
        // the vectors and matrices
        if (_gpuData->imageWidth != imageWidth || _gpuData->imageHeight != imageHeight ||
            _gpuData->isUpsideDown != isUpsideDown || _gpuData->use8Neighbourhood != use8Neighbourhood) {
            // Resize the system vectors and matrices to accomodate the different image size
            _gpuData->imageWidth = imageWidth;
            _gpuData->imageHeight = imageHeight;
            _gpuData->isUpsideDown = isUpsideDown;
            _gpuData->use8Neighbourhood = use8Neighbourhood;
            int numElements = imageWidth * imageHeight;
            int numDiagonals = use8Neighbourhood ? 9 : 5;
            _gpuData->x_h.resize(numElements);
            _gpuData->b_d.resize(numElements);
            _gpuData->x_d.resize(numElements);
            _gpuData->imageBuffer_d.resize(numElements);
            _gpuData->L_d.resize(numElements, numElements, numElements * numDiagonals, numDiagonals);
            _gpuData->L_h.resize(numElements, numElements, numElements * numDiagonals, numDiagonals);
            _gpuData->abFilterR_d.resize(numElements);
            _gpuData->abFilterV_d.resize(numElements);

            // Set the b vector to 0 (except for the row corresponding to the seed points set to 1)
            if (isUpsideDown) {
                CuspDeviceVector::view lastRow(_gpuData->b_d.begin() + (numElements - imageWidth), _gpuData->b_d.end());
                CuspDeviceVector::view rest(_gpuData->b_d.begin(), _gpuData->b_d.begin() + (numElements - imageWidth));
                cusp::blas::fill(rest,    0.0f);
                cusp::blas::fill(lastRow, 1.0f);
            }
            else {
                CuspDeviceVector::view firstRow(_gpuData->b_d.begin(), _gpuData->b_d.begin() + imageWidth);
                CuspDeviceVector::view rest(_gpuData->b_d.begin() + imageWidth, _gpuData->b_d.end());
                cusp::blas::fill(firstRow, 1.0f);
                cusp::blas::fill(rest,     0.0f);
            }

            // Reset x_d to be a linear gradient
            resetSolution();
        }
    }    

    static __device__ float d_getWeight(float v1, float v2, float gradientScaling, float beta, float gamma)
    {
        float grad = abs(v1 - v2) * gradientScaling / 255.0f;
        return exp(-beta * (grad + gamma)) + 1e-4;
    }

    static __global__ void k_buildSystem(float* L, int pitch, const unsigned char* image, int width, int height,
                                  float gradientScaling, float alpha, float beta, float gamma, bool isUpsideDown)
    {

        const int x = blockDim.x * blockIdx.x + threadIdx.x;
        const int y = blockDim.y * blockIdx.y + threadIdx.y;
        const int pidx = y * width + x;
        if (x >= width || y >= height) return;

        const float gamma_sq2 = gamma * 1.4142; // Fixme....
        const int offsets[] = {-width-1, -width, -width+1, -1, 0, 1, width-1, width, width+1};
        const float gammas[] = {gamma_sq2, 0.0f, gamma_sq2, gamma, 0.0f, gamma, gamma_sq2, 0.0f, gamma_sq2};

        // Precompute the three attenuation values for the curernt pixel (the row above, current row, and row below)
        float attenuations[3];
        if (isUpsideDown) {
            attenuations[0] = 1.0f - exp(-alpha * (1.0f - (y - 1.0f) / (height - 1.0f)));
            attenuations[1] = 1.0f - exp(-alpha * (1.0f - (y       ) / (height - 1.0f)));
            attenuations[2] = 1.0f - exp(-alpha * (1.0f - (y + 1.0f) / (height - 1.0f)));
        } else {
            attenuations[0] = 1.0f - exp(-alpha * (y - 1.0f) / (height - 1.0f));
            attenuations[1] = 1.0f - exp(-alpha * (y       ) / (height - 1.0f));
            attenuations[2] = 1.0f - exp(-alpha * (y + 1.0f) / (height - 1.0f));
        }

        // Filter off out-of-bounds edges
        unsigned short filter = 495; // 111 101 111

        // 8 - neighbourhood filter
        if (x == 0)        filter &= 203; // 011 001 011
        if (x == width-1)  filter &= 422; // 110 100 110
        if (y == 0)        filter &=  47; // 000 101 111
        if (y == height-1) filter &= 488; // 111 101 000

        // get central pixel
        float centralValue = image[pidx] * attenuations[1];

        // If the pixel is at the top or at the bottom, add a value of 1 to the diagonal, to
        // account for the edge to the seed points
        float weightSum = 0.0f;
        if (y == 0 || y == height - 1)
            weightSum = 1.0f;

        for (int d = 0; d < 9; ++d) {
            float weight = 0.0f;
            
            if (((256>>d) & filter) != 0) {
                int pidx_2 = pidx + offsets[d];
                float v = image[pidx_2] * attenuations[d/3];
                weight = d_getWeight(centralValue, v, gradientScaling, beta, gammas[d]);
            }

            // The matrix stores the data, so that values on the same diagonal are sequential.
            // This means that all the values from [0, pitch) are on the first diagonal, [pitch, 2*pitch)
            // are on the second diagonal and so on...
            L[d * pitch + pidx] = -weight;
            weightSum += weight;
        }
        L[4 * pitch + pidx] = weightSum;
    }

    // TODO: Unify system creation kernel code for 4 and 8 neighbourhood. DRYness!
    static __global__ void k_buildSystem4Neighbourhood(float* L, int pitch, const unsigned char* image, int width, int height,
                                  float gradientScaling, float alpha, float beta, float gamma, bool isUpsideDown)
    {

        const int x = blockDim.x * blockIdx.x + threadIdx.x;
        const int y = blockDim.y * blockIdx.y + threadIdx.y;
        const int pidx = y * width + x;
        if (x >= width || y >= height) return;

        const int offsets[] = {-width, -1, 0, 1, width};
        const float gammas[] = {0.0f, gamma, 0.0f, gamma, 0.0f};

        // Precompute the three attenuation values for the curernt pixel (the row above, current row, and row below)
        float attenuations[3];
        if (isUpsideDown) {
            attenuations[0] = 1.0f - exp(-alpha * (1.0f - (y - 1.0f) / (height - 1.0f)));
            attenuations[1] = 1.0f - exp(-alpha * (1.0f - (y       ) / (height - 1.0f)));
            attenuations[2] = 1.0f - exp(-alpha * (1.0f - (y + 1.0f) / (height - 1.0f)));
        } else {
            attenuations[0] = 1.0f - exp(-alpha * (y - 1.0f) / (height - 1.0f));
            attenuations[1] = 1.0f - exp(-alpha * (y       ) / (height - 1.0f));
            attenuations[2] = 1.0f - exp(-alpha * (y + 1.0f) / (height - 1.0f));
        }

        // Filter off out-of-bounds edges
        unsigned char filter = 27; // 1 101 1

        // 4 - neighbourhood filter
        if (x == 0)        filter &= 19; // 1 001 1
        if (x == width-1)  filter &= 25; // 1 100 1
        if (y == 0)        filter &= 11; // 0 101 1
        if (y == height-1) filter &= 26; // 1 101 0

        // get central pixel
        float centralValue = image[pidx] * attenuations[1];

        // If the pixel is at the top or at the bottom, add a value of 1 to the diagonal, to
        // account for the edge to the seed points
        float weightSum = 0.0f;
        if (y == 0 || y == height - 1)
            weightSum = 1.0f;

        for (int d = 0; d < 5; ++d) {
            float weight = 0.0f;
            
            if (((16>>d) & filter) != 0) {
                int pidx_2 = pidx + offsets[d];
                float v = image[pidx_2] * attenuations[(d+2)/3];
                weight = d_getWeight(centralValue, v, gradientScaling, beta, gammas[d]);
            }

            // The matrix stores the data, so that values on the same diagonal are sequential.
            // This means that all the values from [0, pitch) are on the first diagonal, [pitch, 2*pitch)
            // are on the second diagonal and so on...
            L[d * pitch + pidx] = -weight;
            weightSum += weight;
        }
        // Store sum of weights in the central diagonal
        L[2 * pitch + pidx] = weightSum;
    }

    void CudaConfidenceMapsSystemSolver::createSystemGPU(const unsigned char* imageData, int imageWidth, int imageHeight,
                                                     float gradientScaling, float alpha, float beta, float gamma,
                                                     bool isUpsideDown) {
        // Initialize the DIA matrix diagonal offsets
        if (_gpuData->use8Neighbourhood) {
            int offsets[9] = {-imageWidth-1, -imageWidth, -imageWidth+1, -1, 0, 1, imageWidth-1, imageWidth, imageWidth+1};
            for (int i = 0; i < 9; ++i) {
                _gpuData->L_d.diagonal_offsets[i] = offsets[i];
            }
        } else {
            int offsets[5] = {-imageWidth, -1, 0, 1, imageWidth};
            for (int i = 0; i < 5; ++i) {
                _gpuData->L_d.diagonal_offsets[i] = offsets[i];
            }
        }

        int numElements = imageWidth * imageHeight;
        dim3 dimBlock(32, 32, 1);
        dim3 dimGrid((imageWidth + 31) / 32, (imageHeight + 31) / 32, 1);

        // Since the image will be needed by the CUDA kernel, it needs to be copied on the GPU first
        hipMemcpy(thrust::raw_pointer_cast(&_gpuData->imageBuffer_d[0]), imageData, numElements, hipMemcpyHostToDevice);
        if (_gpuData->use8Neighbourhood) {   
            k_buildSystem<<<dimGrid, dimBlock>>>(thrust::raw_pointer_cast(&_gpuData->L_d.values.values[0]), static_cast<int>(_gpuData->L_d.values.pitch),
                                                 thrust::raw_pointer_cast(&_gpuData->imageBuffer_d[0]),
                                                 imageWidth, imageHeight,
                                                 gradientScaling, alpha, beta, gamma, _gpuData->isUpsideDown);
        }
        else {
            k_buildSystem4Neighbourhood<<<dimGrid, dimBlock>>>(thrust::raw_pointer_cast(&_gpuData->L_d.values.values[0]), static_cast<int>(_gpuData->L_d.values.pitch),
                                                 thrust::raw_pointer_cast(&_gpuData->imageBuffer_d[0]),
                                                 imageWidth, imageHeight,
                                                 gradientScaling, alpha, beta, gamma, _gpuData->isUpsideDown);            
        }
    }

} // cuda
} // campvis