#include "hip/hip_runtime.h"
#include "cudaconfidencemaps_cuda.h"

#include <cusp/dia_matrix.h>
#include <cusp/krylov/cg.h>
#include <cusp/precond/diagonal.h>

typedef cusp::dia_matrix<int, float, cusp::host_memory> CuspHostDiaMatrix;
typedef cusp::dia_matrix<int, float, cusp::device_memory> CuspDeviceDiaMatrix;
typedef cusp::array1d<float, cusp::host_memory> CuspHostVector;
typedef cusp::array1d<float, cusp::device_memory> CuspDeviceVector;

namespace campvis {
namespace cuda {

    struct CudaConfidenceMapsSystemGPUData {
        // Data on the host (only solution vector x and matrix L)
        CuspHostDiaMatrix L_h;
        CuspHostVector x_h;

        // Data on the device, a.k.a. GPU (Lx = b)
        CuspDeviceDiaMatrix L_d;
        CuspDeviceVector b_d;
        CuspDeviceVector x_d;

        // Information about the system
        bool isUpsideDown;
        int imageWidth;
        int imageHeight;
        float solutionResidualNorm;
    };

    struct ComputeLaplacianData
    {
        float alpha, beta, gamma;
        float gradientScaling;
        const unsigned char *image;
        int width, height;
        int centralDiagonal;
        int offsets[9];
        float gammaList[9];

        std::vector<float> attenuationLUT;
    };

    CudaConfidenceMapsSystemSolver::CudaConfidenceMapsSystemSolver()
        : _gpuData(new CudaConfidenceMapsSystemGPUData())
    {
        _gpuData->imageWidth = 0;
        _gpuData->imageHeight = 0;
        _gpuData->solutionResidualNorm = 0.0f;
    }

    static inline float _getWeight(const ComputeLaplacianData &data, int x, int y, int diagonal)
    {
        const unsigned char *image = data.image;

        int idx1 = y * data.width + x;
        int idx2 = idx1 + data.offsets[diagonal];

        float attenuation1 = data.attenuationLUT[idx1 / data.width];
        float attenuation2 = data.attenuationLUT[idx2 / data.width];

        float gradient = abs(image[idx1]*attenuation1/255.0f - image[idx2]*attenuation2/255.0f) * data.gradientScaling;

        float weight = exp(-data.beta * (gradient + data.gammaList[diagonal]));
        return weight + 1e-4;
    }

    void CudaConfidenceMapsSystemSolver::uploadImage(const unsigned char* imageData, int imageWidth, int imageHeight,
                                                     float gradientScaling, float alpha, float beta, float gamma,
                                                     bool isUpsideDown) {
        resizeDataStructures(imageWidth, imageHeight, isUpsideDown);

        // Gather all the parameters needed to create the system in one place
        ComputeLaplacianData data;
        data.alpha = alpha;
        data.beta = beta;
        data.gamma = gamma;
        data.gradientScaling = gradientScaling;
        data.image = imageData;
        data.width = imageWidth;
        data.height = imageHeight;
        data.centralDiagonal = 4;
        int offsets[9] = {-imageWidth-1, -imageWidth, -imageWidth+1, -1, 0, 1, imageWidth-1, imageWidth, imageWidth+1};
        float gammaList[9] = {sqrt(2.0f)*gamma, 0.0f, sqrt(2.0f)*gamma, gamma, 0.0f, gamma, sqrt(2.0f)*gamma, 0.0f, sqrt(2.0f)*gamma};

        for (int i = 0; i < 9; ++i) {
            data.offsets[i] = offsets[i];
            data.gammaList[i] = gammaList[i];
            _gpuData->L_h.diagonal_offsets[i] = offsets[i];
        }

        // Precompute attenuation tables
        data.attenuationLUT = std::vector<float>(imageHeight);
        for (int i = 0; i < imageHeight; ++i) {
            float y = (float)i / (float)(imageHeight-1);
            if (isUpsideDown) y = 1 - y;
            data.attenuationLUT[i] = 1 - exp(-alpha * y);
        }

        // Fill in the rest of the matrix
        for (int y = 0; y < imageHeight; ++y) {
            for (int x = 0; x < imageWidth; ++x) {
                int idx = y * imageWidth + x;

                // Filter off out-of-bounds edges
                unsigned short filter = 495; // 111 101 111

                // 8 - neighbourhood filter
                if (x == 0)        filter &= 203; // 011 001 011
                if (x == imageWidth-1)  filter &= 422; // 110 100 110
                if (y == 0)        filter &=  47; // 000 101 111
                if (y == imageHeight-1) filter &= 488; // 111 101 000

                float valueSum = 0.0f;
                if (y == 0 || y == imageHeight - 1) valueSum = 1.0f;

                for (int d = 0; d < 9; ++d) {
                    _gpuData->L_h.values(idx, d) = 0;

                    float value = 0.0f;
                    
                    if (((256>>d) & filter) != 0) {
                        value = _getWeight(data, x, y, d);
                        _gpuData->L_h.values(idx, d) = -value;
                    }

                    valueSum += value;
                }

                _gpuData->L_h.values(idx, data.centralDiagonal) = valueSum;
            }
        }

        // Upload system
        _gpuData->L_d = _gpuData->L_h;
    }

    void CudaConfidenceMapsSystemSolver::resetSolution() {
        int width = _gpuData->imageWidth;
        int height = _gpuData->imageHeight;
        bool isUpsideDown = _gpuData->isUpsideDown;

        for (int y = 0; y < height; ++y) {
            for (int x = 0; x < width; ++x) {
                float value;
                if (!isUpsideDown)
                    value = y / (height - 1.0f);
                else
                    value = 1.0f - y / (height - 1.0f);

                _gpuData->x_h[y * width + x] = value;
            }
        }

        // Uplaod solution vector on the GPU
        _gpuData->x_d = _gpuData->x_h;
    }

    void CudaConfidenceMapsSystemSolver::solve(int maximumIterations, float errorTolerance) {
        cusp::default_monitor<float> monitor(_gpuData->b_d, maximumIterations, errorTolerance);
        cusp::precond::diagonal<float, cusp::device_memory> M(_gpuData->L_d);
        cusp::krylov::cg(_gpuData->L_d, _gpuData->x_d, _gpuData->b_d, monitor, M);

        _gpuData->solutionResidualNorm = monitor.residual_norm();
    }

    const float* CudaConfidenceMapsSystemSolver::getSolution(int& width, int& height) {
        // Downlaod data
        _gpuData->x_h = _gpuData->x_d;
        return &_gpuData->x_h[0];
    }

    float CudaConfidenceMapsSystemSolver::getSolutionResidualNorm() const {
        return _gpuData->solutionResidualNorm;
    }

    void CudaConfidenceMapsSystemSolver::resizeDataStructures(int imageWidth, int imageHeight, bool isUpsideDown) {
        // If the problem size changed, reset the solution vector, as well as resizing the vectors and matrices
        if (_gpuData->imageWidth != imageWidth || _gpuData->imageHeight != imageHeight || _gpuData->isUpsideDown != isUpsideDown) {
            // Resize the system vectors and matrices to accomodate the different image sze
            _gpuData->imageWidth = imageWidth;
            _gpuData->imageHeight = imageHeight;
            int numElements = imageWidth * imageHeight;
            _gpuData->x_h.resize(numElements);
            _gpuData->b_d.resize(numElements);
            _gpuData->x_d.resize(numElements);
            _gpuData->L_d.resize(numElements, numElements, numElements * 9, 9);
            _gpuData->L_h.resize(numElements, numElements, numElements * 9, 9);

            // Set the b vector to 0 (except for the row corresponding to the seed points set to 1)
            if (isUpsideDown) {
                CuspDeviceVector::view lastRow(_gpuData->b_d.begin() + (numElements - imageWidth), _gpuData->b_d.end());
                CuspDeviceVector::view rest(_gpuData->b_d.begin(), _gpuData->b_d.begin() + (numElements - imageWidth));
                cusp::blas::fill(rest,    0.0f);
                cusp::blas::fill(lastRow, 1.0f);
            }
            else {
                CuspDeviceVector::view firstRow(_gpuData->b_d.begin(), _gpuData->b_d.begin() + imageWidth);
                CuspDeviceVector::view rest(_gpuData->b_d.begin() + imageWidth, _gpuData->b_d.end());
                cusp::blas::fill(firstRow, 1.0f);
                cusp::blas::fill(rest,     0.0f);
            }

            resetSolution();
        }
    }

} // cuda
} // campvis