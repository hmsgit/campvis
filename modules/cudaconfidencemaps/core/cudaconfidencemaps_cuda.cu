#include "hip/hip_runtime.h"
#include "cudaconfidencemaps_cuda.h"

#include <cusp/dia_matrix.h>
#include <cusp/krylov/cg.h>
#include <cusp/precond/diagonal.h>

typedef cusp::dia_matrix<int, float, cusp::host_memory> CuspHostDiaMatrix;
typedef cusp::dia_matrix<int, float, cusp::device_memory> CuspDeviceDiaMatrix;
typedef cusp::array1d<float, cusp::host_memory> CuspHostVector;
typedef cusp::array1d<float, cusp::device_memory> CuspDeviceVector;

#define CUDACONFIDENCEMAPS_USE_GPU_FOR_SYSTEM_CREATION 1;

namespace campvis {
namespace cuda {

    struct CudaConfidenceMapsSystemGPUData {
        // Data on the host (only solution vector x and matrix L)
        CuspHostDiaMatrix L_h;
        CuspHostVector x_h;

        // Data on the device, a.k.a. GPU (Lx = b)
        CuspDeviceDiaMatrix L_d;
        CuspDeviceVector b_d;
        CuspDeviceVector x_d;

        // Information about the system
        bool isUpsideDown;
        int imageWidth;
        int imageHeight;
        float solutionResidualNorm;
        float systemCreationTime;
        float systemSolveTime;
    };

    struct ComputeLaplacianData
    {
        float alpha, beta, gamma;
        float gradientScaling;
        const unsigned char *image;
        int width, height;
        int centralDiagonal;
        int offsets[9];
        float gammaList[9];

        std::vector<float> attenuationLUT;
    };

    CudaConfidenceMapsSystemSolver::CudaConfidenceMapsSystemSolver()
        : _gpuData(new CudaConfidenceMapsSystemGPUData())
    {
        _gpuData->imageWidth = 0;
        _gpuData->imageHeight = 0;
        _gpuData->solutionResidualNorm = 0.0f;
    }

    void CudaConfidenceMapsSystemSolver::uploadImage(const unsigned char* imageData, int imageWidth, int imageHeight,
                                                     float gradientScaling, float alpha, float beta, float gamma,
                                                     bool useGPU, bool isUpsideDown) {
        resizeDataStructures(imageWidth, imageHeight, isUpsideDown);

        // Measure execution time
        hipEvent_t start, stop;
        hipEventCreate(&start);
        hipEventCreate(&stop);
        hipEventRecord(start, 0);

        if (useGPU) {
            createSystemGPU(imageData, imageWidth, imageHeight, gradientScaling, alpha, beta, gamma, isUpsideDown);
        } else {
            createSystemCPU(imageData, imageWidth, imageHeight, gradientScaling, alpha, beta, gamma, isUpsideDown);
        }

        hipEventRecord(stop, 0);
        hipEventSynchronize(stop);
        hipEventElapsedTime(&_gpuData->systemCreationTime, start, stop);

    }

    void CudaConfidenceMapsSystemSolver::resetSolution() {
        int width = _gpuData->imageWidth;
        int height = _gpuData->imageHeight;
        bool isUpsideDown = _gpuData->isUpsideDown;

        for (int y = 0; y < height; ++y) {
            for (int x = 0; x < width; ++x) {
                float value;
                if (!isUpsideDown)
                    value = y / (height - 1.0f);
                else
                    value = 1.0f - y / (height - 1.0f);

                _gpuData->x_h[y * width + x] = value;
            }
        }

        // Uplaod solution vector on the GPU
        _gpuData->x_d = _gpuData->x_h;
    }

    void CudaConfidenceMapsSystemSolver::solve(int maximumIterations, float errorTolerance) {
        // Measure execution time
        hipEvent_t start, stop;
        hipEventCreate(&start);
        hipEventCreate(&stop);
        hipEventRecord(start, 0);

        cusp::default_monitor<float> monitor(_gpuData->b_d, maximumIterations, errorTolerance);
        cusp::precond::diagonal<float, cusp::device_memory> M(_gpuData->L_d);
        cusp::krylov::cg(_gpuData->L_d, _gpuData->x_d, _gpuData->b_d, monitor, M);

        // Downlaod data
        _gpuData->x_h = _gpuData->x_d;

        _gpuData->solutionResidualNorm = monitor.residual_norm();

        hipEventRecord(stop, 0);
        hipEventSynchronize(stop);
        hipEventElapsedTime(&_gpuData->systemSolveTime, start, stop);        
    }

    const float* CudaConfidenceMapsSystemSolver::getSolution(int& width, int& height) {
        width = _gpuData->imageWidth;
        height = _gpuData->imageHeight;
        return &_gpuData->x_h[0];
    }

    float CudaConfidenceMapsSystemSolver::getSolutionResidualNorm() const {
        return _gpuData->solutionResidualNorm;
    }

    float CudaConfidenceMapsSystemSolver::getSystemCreationTime() const {
        return _gpuData->systemCreationTime;
    }
    
    float CudaConfidenceMapsSystemSolver::getSystemSolveTime() const {
        return _gpuData->systemSolveTime;
    }


    void CudaConfidenceMapsSystemSolver::resizeDataStructures(int imageWidth, int imageHeight, bool isUpsideDown) {
        // If the problem size changed, reset the solution vector, as well as resizing the vectors and matrices
        if (_gpuData->imageWidth != imageWidth || _gpuData->imageHeight != imageHeight || _gpuData->isUpsideDown != isUpsideDown) {
            // Resize the system vectors and matrices to accomodate the different image sze
            _gpuData->imageWidth = imageWidth;
            _gpuData->imageHeight = imageHeight;
            int numElements = imageWidth * imageHeight;
            _gpuData->x_h.resize(numElements);
            _gpuData->b_d.resize(numElements);
            _gpuData->x_d.resize(numElements);
            _gpuData->L_d.resize(numElements, numElements, numElements * 9, 9);
            _gpuData->L_h.resize(numElements, numElements, numElements * 9, 9);

            // Set the b vector to 0 (except for the row corresponding to the seed points set to 1)
            if (isUpsideDown) {
                CuspDeviceVector::view lastRow(_gpuData->b_d.begin() + (numElements - imageWidth), _gpuData->b_d.end());
                CuspDeviceVector::view rest(_gpuData->b_d.begin(), _gpuData->b_d.begin() + (numElements - imageWidth));
                cusp::blas::fill(rest,    0.0f);
                cusp::blas::fill(lastRow, 1.0f);
            }
            else {
                CuspDeviceVector::view firstRow(_gpuData->b_d.begin(), _gpuData->b_d.begin() + imageWidth);
                CuspDeviceVector::view rest(_gpuData->b_d.begin() + imageWidth, _gpuData->b_d.end());
                cusp::blas::fill(firstRow, 1.0f);
                cusp::blas::fill(rest,     0.0f);
            }

            resetSolution();
        }
    }

    static inline float _getWeight(const ComputeLaplacianData &data, int x, int y, int diagonal)
    {
        const unsigned char *image = data.image;

        int idx1 = y * data.width + x;
        int idx2 = idx1 + data.offsets[diagonal];

        float attenuation1 = data.attenuationLUT[idx1 / data.width];
        float attenuation2 = data.attenuationLUT[idx2 / data.width];

        float gradient = abs(image[idx1]*attenuation1/255.0f - image[idx2]*attenuation2/255.0f) * data.gradientScaling;

        float weight = exp(-data.beta * (gradient + data.gammaList[diagonal]));
        return weight + 1e-4;
    }

    void CudaConfidenceMapsSystemSolver::createSystemCPU(const unsigned char* imageData, int imageWidth, int imageHeight,
                                                        float gradientScaling, float alpha, float beta, float gamma,
                                                        bool isUpsideDown) {
        // Gather all the parameters needed to create the system in one place
        ComputeLaplacianData data;
        data.alpha = alpha;
        data.beta = beta;
        data.gamma = gamma;
        data.gradientScaling = gradientScaling;
        data.image = imageData;
        data.width = imageWidth;
        data.height = imageHeight;
        data.centralDiagonal = 4;
        int offsets[9] = {-imageWidth-1, -imageWidth, -imageWidth+1, -1, 0, 1, imageWidth-1, imageWidth, imageWidth+1};
        float gammaList[9] = {sqrt(2.0f)*gamma, 0.0f, sqrt(2.0f)*gamma, gamma, 0.0f, gamma, sqrt(2.0f)*gamma, 0.0f, sqrt(2.0f)*gamma};

        for (int i = 0; i < 9; ++i) {
            data.offsets[i] = offsets[i];
            data.gammaList[i] = gammaList[i];
            _gpuData->L_h.diagonal_offsets[i] = offsets[i];
            _gpuData->L_d.diagonal_offsets[i] = offsets[i];
        }

        // Precompute attenuation tables
        data.attenuationLUT = std::vector<float>(imageHeight);
        for (int i = 0; i < imageHeight; ++i) {
            float y = (float)i / (float)(imageHeight-1);
            if (isUpsideDown) y = 1 - y;
            data.attenuationLUT[i] = 1 - exp(-alpha * y);
        }

        // Fill in the rest of the matrix
        for (int y = 0; y < imageHeight; ++y) {
            for (int x = 0; x < imageWidth; ++x) {
                int idx = y * imageWidth + x;

                // Filter off out-of-bounds edges
                unsigned short filter = 495; // 111 101 111

                // 8 - neighbourhood filter
                if (x == 0)        filter &= 203; // 011 001 011
                if (x == imageWidth-1)  filter &= 422; // 110 100 110
                if (y == 0)        filter &=  47; // 000 101 111
                if (y == imageHeight-1) filter &= 488; // 111 101 000

                float valueSum = 0.0f;
                if (y == 0 || y == imageHeight - 1) valueSum = 1.0f;

                for (int d = 0; d < 9; ++d) {
                    float value = 0.0f;
                    
                    if (((256>>d) & filter) != 0) {
                        value = _getWeight(data, x, y, d);
                    }

                    _gpuData->L_h.values(idx, d) = -value;
                    valueSum += value;
                }

                _gpuData->L_h.values(idx, data.centralDiagonal) = valueSum;
            }
        }

        // Upload system
        _gpuData->L_d = _gpuData->L_h;
    }

    static __device__ float d_getWeight(float v1, float v2, float gradientScaling, float beta, float gamma)
    {
        float grad = abs(v1 - v2) * gradientScaling / 255.0f;
        return exp(-beta * (grad + gamma)) + 1e-4;
    }

    static __global__ void k_buildSystem(float* L, int pitch, const unsigned char* image, int width, int height,
                                  float gradientScaling, float alpha, float beta, float gamma)
    {

        const int x = blockDim.x * blockIdx.x + threadIdx.x;
        const int y = blockDim.y * blockIdx.y + threadIdx.y;
        const int pidx = y * width + x;
        if (x >= width || y >= height) return;

        const float gamma_sq2 = gamma * 1.4142; // Fixme....
        const int offsets[] = {-width-1, -width, -width+1, -1, 0, 1, width-1, width, width+1};
        const float gammas[] = {gamma_sq2, 0.0f, gamma_sq2, gamma, 0.0f, gamma, gamma_sq2, 0.0f, gamma_sq2};

        const float attenuations[] = {
            1.0f - (y - 1.0f) / (height - 1.0f),
            1.0f - (y       ) / (height - 1.0f),
            1.0f - (y + 1.0f) / (height - 1.0f)
        };

        // Filter off out-of-bounds edges
        unsigned short filter = 495; // 111 101 111

        // 8 - neighbourhood filter
        if (x == 0)        filter &= 203; // 011 001 011
        if (x == width-1)  filter &= 422; // 110 100 110
        if (y == 0)        filter &=  47; // 000 101 111
        if (y == height-1) filter &= 488; // 111 101 000

        // get central pixel
        float centralValue = image[pidx] * attenuations[1];

        // If the pixel is at the top or at the bottom, add a value of 1 to the diagonal, to
        // account for the edge to the seed points
        float valueSum = 0.0f;
        if (y == 0 || y == height - 1)
            valueSum = 1.0f;

        for (int d = 0; d < 9; ++d) {
            float value = 0.0f;
            
            if (((256>>d) & filter) != 0) {
                int pidx_2 = pidx + offsets[d];
                float v = image[pidx_2] * attenuations[d/3];
                value = d_getWeight(centralValue, v, gradientScaling, beta, gammas[d]);
            }

            L[d * pitch + pidx] = -value;
            valueSum += value;
        }
        L[4 * pitch + pidx] = valueSum;
    }

    void CudaConfidenceMapsSystemSolver::createSystemGPU(const unsigned char* imageData, int imageWidth, int imageHeight,
                                                     float gradientScaling, float alpha, float beta, float gamma,
                                                     bool isUpsideDown) {

        int offsets[9] = {-imageWidth-1, -imageWidth, -imageWidth+1, -1, 0, 1, imageWidth-1, imageWidth, imageWidth+1};

        for (int i = 0; i < 9; ++i) {
            _gpuData->L_d.diagonal_offsets[i] = offsets[i];
        }

        int numElements = imageWidth * imageHeight;

        dim3 dimBlock(32, 32, 1);
        dim3 dimGrid((imageWidth + 31) / 32, (imageHeight + 31) / 32, 1);
        unsigned char *deviceImage;
        hipMalloc((void**)&deviceImage, numElements);
        hipMemcpy(deviceImage, imageData, numElements, hipMemcpyHostToDevice);
        k_buildSystem<<<dimGrid, dimBlock>>>(thrust::raw_pointer_cast(&_gpuData->L_d.values.values[0]), _gpuData->L_d.values.pitch,
                                             deviceImage, imageWidth, imageHeight,
                                             gradientScaling, alpha, beta, gamma);
        hipFree(deviceImage);
    }

} // cuda
} // campvis