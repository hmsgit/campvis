
#include <hip/hip_runtime.h>

/*<

__global__ void k_buildCMSystem(cudaTextureObject_t image, int width, int height,
                                float alpha, float beta, float gamma,
                                float *out_weights)
{
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;
	int idx = y * width + x;

	if (x >= width || y >= height) return;

	float attenuation_center = exp(-alpha * (float)y / height);
	float attenuation_bottom = exp(-alpha * (float)(y+1) / height);

	float c_center = tex2D(image, x, y)     * attenuation_center;
	float c_right  = tex2D(image, x+1, y)   * attenuation_center;
	float c_bottom = tex2D(image, x, y+1)   * attenuation_bottom;
	float c_se     = tex2D(image, x+1, y+1) * attenuation_bottom;
	float c_sw     = tex2D(image, x-1, y+1) * attenuation_bottom;

	float w_right  = exp(-beta * (abs(c_center - c_right) + gamma));
	float w_bottom = exp(-beta * (abs(c_center - c_bottom)));
	float w_se     = exp(-beta * (abs(c_center - c_se) + sqrt(2.0f) * gamma));
	float w_sw     = exp(-beta * (abs(c_center - c_sw) + sqrt(2.0f) * gamma));

	out_weights[idx*4 + 0] = w_right;
	out_weights[idx*4 + 1] = w_sw;
	out_weights[idx*4 + 2] = w_bottom;
	out_weights[idx*4 + 3] = w_se;
}


int main()
{
	// Create texture object
	cudaResourceDesc resDesc;
	memset(&resDesc, 0, sizeof(resDesc));
	resDesc.resType = cudaResourceTypePitch2D


	return 0;
}*/