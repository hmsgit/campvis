#include "hip/hip_runtime.h"
#include <vector>
#include <cmath>
#include <cusp/dia_matrix.h>
#include <cusp/krylov/cg.h>
#include <cusp/precond/diagonal.h>

#include "cm_cusp_cuda_exports.h"

typedef cusp::dia_matrix<int, float, cusp::host_memory> CuspHostDiaMatrix;
typedef cusp::dia_matrix<int, float, cusp::device_memory> CuspDeviceDiaMatrix;
typedef cusp::array1d<float, cusp::host_memory> CuspHostVector;
typedef cusp::array1d<float, cusp::device_memory> CuspDeviceVector;

class CuspGPUData {
public:
	CuspGPUData(int width, int height)
		: L_h(width*height, width*height, width*height*9, 9),
	      b_h(width*height), x_h(width*height),
		  L_d(width*height, width*height, width*height*9, 9),
	      b_d(width*height), x_d(width*height) {};

	void resize(int width, int height)
	{
		this->L_h.resize(width*height, width*height, width*height*9, 9);
		this->b_h.resize(width*height);
		this->x_h.resize(width*height);
	}

	CuspHostDiaMatrix L_h;
	CuspHostVector b_h;
	CuspHostVector x_h;

	CuspDeviceDiaMatrix L_d;
	CuspDeviceVector b_d;
	CuspDeviceVector x_d;
};

CuspGPUData* CUSP_CM_createGpuData(int width, int height)
{
	CuspGPUData *data = new CuspGPUData(width, height);
	return data;
}

void CUSP_CM_destroyGpuData(CuspGPUData *data)
{
	delete data;
}

void CUSP_CM_resizeGpuData(CuspGPUData& data, int width, int height)
{
	data.resize(width, height);
}

void CUSP_CM_setInitialSolution(CuspGPUData& data, const std::vector<float>& values)
{
	for (int i = 0; i < data.x_h.size() && i < values.size(); ++i) {
		data.x_h[i] = values[i];
	}
}

void CUSP_CM_uploadSystem(CuspGPUData &data)
{
	data.x_d = data.x_h;
	data.b_d = data.b_h;
	data.L_d = data.L_h;
}

void CUSP_CM_downloadSolution(CuspGPUData &data)
{
	data.x_h = data.x_d;
	// FIXME: Should not be needed... (Clamp solution)
	for (int i = 0; i < data.x_h.size(); ++i) {
		data.x_h[i] = min(1.0f, data.x_h[i]);
	}
}

void CUSP_CM_solveSystem(CuspGPUData& data, int iterations, float precision)
{
	cusp::default_monitor<float> monitor(data.b_d, iterations, precision);
	//cusp::precond::bridson_ainv<float, cusp::device_memory> M(A, 1);
	cusp::precond::diagonal<float, cusp::device_memory> M(data.L_d);
	//cusp::krylov::bicgstab(A, x, b, monitor);
	cusp::krylov::cg(data.L_d, data.x_d, data.b_d, monitor, M);
}

const float* CUSP_CM_getSolutionPtr(const CuspGPUData& data)
{
	//const_cast<CuspGPUData&>(data).x_h = const_cast<CuspGPUData&>(data).x_d;
	return &data.x_h[0];
}




struct ComputeLaplacianData
{
	float alpha, beta, gamma;
	float gradientScaling;
	const unsigned char *image;
	int width, height;
	int centralDiagonal;
	int offsets[9];
	float gammaList[9];

	std::vector<float> attenuationLUT;
};

static inline float _getWeight(const ComputeLaplacianData &data, int x, int y, int diagonal, bool isUpsideDown)
{
	const unsigned char *image = data.image;

	int idx1 = y * data.width + x;
	int idx2 = idx1 + data.offsets[diagonal];

	float attenuation1 = data.attenuationLUT[idx1 / data.width];
	float attenuation2 = data.attenuationLUT[idx2 / data.width];

	float gradient = abs(image[idx1]*attenuation1/255.0f - image[idx2]*attenuation2/255.0f) * data.gradientScaling;

	float weight = exp(-data.beta * (gradient + data.gammaList[diagonal]));
	return weight + 1e-4;
}

void CUSP_CM_buildEquationSystem(CuspGPUData &gpudata, const unsigned char* image, int width, int height,
                                 float alpha, float beta, float gamma,
                                 float gradientScaling,
                                 bool isUpsideDown)
{
	// Gather all of the options together
	ComputeLaplacianData data;
	data.alpha = alpha; data.beta = beta; data.gamma = gamma;
	data.gradientScaling = gradientScaling;
	data.image = image;
	data.width = width; data.height = height;
	data.centralDiagonal = 4;
	int offsets[9] = {-width-1, -width, -width+1, -1, 0, 1, width-1, width, width+1};
	float gammaList[9] = {sqrt(2.0f)*gamma, 0.0f, sqrt(2.0f)*gamma, gamma, 0.0f, gamma, sqrt(2.0f)*gamma, 0.0f, sqrt(2.0f)*gamma};
	for (int i = 0; i < 9; ++i) {
		data.offsets[i] = offsets[i];
		data.gammaList[i] = gammaList[i];
	}

	// Prepare equation system data structure
	for (int i = 0; i < 9; ++i) {
		gpudata.L_h.diagonal_offsets[i] = data.offsets[i];
	}

	// Precompute attenuation tables
	data.attenuationLUT = std::vector<float>(height);
	for (int i = 0; i < height; ++i) {
		float y = (float)i / (float)(height-1);
		if (isUpsideDown) y = 1 - y;
		data.attenuationLUT[i] = 1 - exp(-alpha * y);
	}

	// Initialize B
	for (int x = 0; x < width * height; ++x) {
		if (x < width)
			gpudata.b_h[x] = isUpsideDown ? 0.0f : 1.0f;
		else if (x >= (width*(height-1)))
			gpudata.b_h[x] = isUpsideDown ? 1.0f : 0.0f;
		else
			gpudata.b_h[x] = 0.0f;
	}

	// Fill in the rest of the matrix
	for (int y = 0; y < height; ++y) {
		for (int x = 0; x < width; ++x) {
			int idx = y * width + x;

			// Filter off out-of-bounds edges
			unsigned short filter = 495; // 111 101 111

			// 8 - neighbourhood filter
			if (x == 0)        filter &= 203; // 011 001 011
			if (x == width-1)  filter &= 422; // 110 100 110
			if (y == 0)        filter &=  47; // 000 101 111
			if (y == height-1) filter &= 488; // 111 101 000

			// 4 - neighbourhood filter
			//if (x == 0)        filter &= 138; // 010 001 010
			//if (x == width-1)  filter &= 162; // 010 100 010
			//if (y == 0)        filter &=  42; // 000 101 010
			//if (y == height-1) filter &= 168; // 010 101 000

			float valueSum = 0.0f;
			if (y == 0 || y == height - 1) valueSum = 1.0f;

			for (int d = 0; d < 9; ++d) {
				gpudata.L_h.values(idx, d) = 0;

				float value = 0.0f;
				
				if (((256>>d) & filter) != 0) {
					value = _getWeight(data, x, y, d, isUpsideDown);
					gpudata.L_h.values(idx, d) = -value;
				}

				valueSum += value;
			}

			gpudata.L_h.values(idx, data.centralDiagonal) = valueSum;
		}
	}
}