#include "hip/hip_runtime.h"
#include <vector>
#include <cmath>
#include <cusp/dia_matrix.h>
#include <cusp/krylov/cg.h>
#include <cusp/precond/diagonal.h>

#include "cm_cusp_cuda_exports.h"

typedef cusp::dia_matrix<int, float, cusp::host_memory> CuspHostDiaMatrix;
typedef cusp::dia_matrix<int, float, cusp::device_memory> CuspDeviceDiaMatrix;
typedef cusp::array1d<float, cusp::host_memory> CuspHostVector;
typedef cusp::array1d<float, cusp::device_memory> CuspDeviceVector;

class CuspGPUData {
public:
	CuspGPUData(int width, int height)
		: L_h(width*height, width*height, width*height*9, 9),
	      b_h(width*height), x_h(width*height),
		  L_d(width*height, width*height, width*height*9, 9),
	      b_d(width*height), x_d(width*height) {};

	void resize(int width, int height)
	{
		this->L_h.resize(width*height, width*height, width*height*9, 9);
		this->b_h.resize(width*height);
		this->x_h.resize(width*height);
	}

	CuspHostDiaMatrix L_h;
	CuspHostVector b_h;
	CuspHostVector x_h;

	CuspDeviceDiaMatrix L_d;
	CuspDeviceVector b_d;
	CuspDeviceVector x_d;
};

CuspGPUData* CUSP_CM_createGpuData(int width, int height)
{
	CuspGPUData *data = new CuspGPUData(width, height);
	return data;
}

void CUSP_CM_destroyGpuData(CuspGPUData *data)
{
	delete data;
}

void CUSP_CM_resizeGpuData(CuspGPUData& data, int width, int height)
{
	data.resize(width, height);
}

void CUSP_CM_setInitialSolution(CuspGPUData& data, const std::vector<float>& values)
{
	for (int i = 0; i < data.x_h.size() && i < values.size(); ++i) {
		data.x_h[i] = values[i];
	}
}

void CUSP_CM_uploadSystem(CuspGPUData &data)
{
	data.x_d = data.x_h;
	data.b_d = data.b_h;
	data.L_d = data.L_h;
}

void CUSP_CM_downloadSolution(CuspGPUData &data)
{
	data.x_h = data.x_d;
	// FIXME: Should not be needed... (Clamp solution)
	for (int i = 0; i < data.x_h.size(); ++i) {
		data.x_h[i] = min(1.0f, data.x_h[i]);
	}
}

void CUSP_CM_solveSystem(CuspGPUData& data, int iterations, float precision)
{
	cusp::default_monitor<float> monitor(data.b_d, iterations, precision);
	//cusp::precond::bridson_ainv<float, cusp::device_memory> M(A, 1);
	cusp::precond::diagonal<float, cusp::device_memory> M(data.L_d);
	//cusp::krylov::bicgstab(A, x, b, monitor);
	cusp::krylov::cg(data.L_d, data.x_d, data.b_d, monitor, M);
}

const float* CUSP_CM_getSolutionPtr(const CuspGPUData& data)
{
	//const_cast<CuspGPUData&>(data).x_h = const_cast<CuspGPUData&>(data).x_d;
	return &data.x_h[0];
}




struct ComputeLaplacianData
{
	float alpha, beta, gamma;
	float gradientScaling;
	const unsigned char *image;
	int width, height;
	int centralDiagonal;
	int offsets[9];
	float gammaList[9];
};


static inline float _getAttenuation(int y, int height, float alpha) {
	return (1 - exp(-alpha * ((float)y / (float)(height-1))));
}

static inline float _getGradient(const ComputeLaplacianData &data, int idx, int offset)
{
	const unsigned char *image = data.image;
	int y1 = idx / data.width;
	int y2 = (idx+offset) / data.width;

	float a1 = _getAttenuation(y1, data.height, data.alpha);
	float a2 = _getAttenuation(y2, data.height, data.alpha);

	return abs(image[idx]*a1/255.0f - image[idx+offset]*a2/255.0f);
}

static inline float _calculateWeight(float gradient, float beta, float gamma, float scaling)
{
	return exp(-beta * (gradient*scaling + gamma));
}

static inline float _getWeight(const ComputeLaplacianData &data, int x, int y, int diagonal)
{
	float gradient = _getGradient(data, y * data.width + x, data.offsets[diagonal]);
	float weight = _calculateWeight(gradient, data.beta, data.gammaList[diagonal], data.gradientScaling);
	return weight + 1e-4;
}

void CUSP_CM_buildEquationSystem(CuspGPUData &gpudata, const unsigned char* image, int width, int height,
                                 float alpha, float beta, float gamma,
                                 float gradientScaling)
{
	// Gather all of the options together
	ComputeLaplacianData data;
	data.alpha = alpha; data.beta = beta; data.gamma = gamma;
	data.gradientScaling = gradientScaling;
	data.image = image;
	data.width = width; data.height = height;
	data.centralDiagonal = 4;
	int offsets[9] = {-width-1, -width, -width+1, -1, 0, 1, width-1, width, width+1};
	float gammaList[9] = {sqrt(2.0f)*gamma, 0.0f, sqrt(2.0f)*gamma, gamma, 0.0f, gamma, sqrt(2.0f)*gamma, 0.0f, sqrt(2.0f)*gamma};
	for (int i = 0; i < 9; ++i) {
		data.offsets[i] = offsets[i];
		data.gammaList[i] = gammaList[i];
	}

	// Prepare equation system data structure
	for (int i = 0; i < 9; ++i) {
		gpudata.L_h.diagonal_offsets[i] = data.offsets[i];
	}

	// Reset B
	for (int x = 0; x < width * height; ++x) {
		gpudata.b_h[x] = 0.0f;
	}

	// Fill in first row of b, which sholud be one
	for (int x = 0; x < width; ++x) {
		gpudata.b_h[x] = 1.0f;
	}

	// Fill in rows of A corresponding to first and last row of the image
	for (int x = 0; x < width; ++x) {
		for (int d = 0; d < 9; ++d) {
			gpudata.L_h.values(x, d) = (d == data.centralDiagonal ? 1.0f : 0.0f);
			gpudata.L_h.values(width*(height-1) + x, d) = (d == data.centralDiagonal ? 1.0f : 0.0f);
		}
	}

	// Fill in the rest of the matrix
	for (int y = 1; y < height-1; ++y) {
		for (int x = 0; x < width; ++x) {
			int idx = y * width + x;

			// Filter off out-of-bounds edges
			unsigned short filter = 495; // 111 101 111

			if (x == 0)        filter &= 203; // 011 001 011
			if (x == width-1)  filter &= 422; // 110 100 110
			if (y == 1)        filter &=  47; // 000 101 111
			if (y == height-2) filter &= 488; // 111 101 000

			float valueSum = 0.0f;

			for (int d = 0; d < 9; ++d) {
				gpudata.L_h.values(idx, d) = 0;

				float value = 0.0f;
				
				if (((256>>d) & filter) != 0) {
					value = _getWeight(data, x, y, d);
					gpudata.L_h.values(idx, d) = -value;
				} else if(y == 1 || y == height - 2) {
					unsigned short filter2 = 495; // 111 101 11
					if (x == 0)        filter2 &= 203; // 011 001 011
					if (x == width-1)  filter2 &= 422; // 110 100 110
					if (((256>>d) & filter2) != 0) {
						value = _getWeight(data, x, y, d);

						if (y == 1) gpudata.b_h[idx] += value;
					}
				}

				valueSum += value;
			}

			gpudata.L_h.values(idx, data.centralDiagonal) = valueSum;
		}
	}
}